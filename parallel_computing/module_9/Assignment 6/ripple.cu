#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel( unsigned char *ptr, int ticks ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // now calculate the value at that position
    float fx = x - DIM/2;
    float fy = y - DIM/2;
    float d = sqrtf( fx * fx + fy * fy );
    
    // Calculate hue
    float hue = fmodf(d/20.f + ticks/50.0f, 1.0f);

    /// Calculate RGB values using an offset of hue
    unsigned char r = (unsigned char)(sin(6.28318f*hue) * 127.5f+127.5f);
    unsigned char g = (unsigned char)(sin(6.28318f*hue+2.09439f) * 127.5f+127.5f);
    unsigned char b = (unsigned char)(sin(6.28318f*hue+4.18879f) * 127.5f+127.5f);

    // Assign RGB values to a pointer
    ptr[offset*4 + 0] = r;
    ptr[offset*4 + 1] = g;
    ptr[offset*4 + 2] = b;
    ptr[offset*4 + 3] = 255;
}

struct DataBlock {
    unsigned char   *dev_bitmap;
    CPUAnimBitmap  *bitmap;
};

void generate_frame( DataBlock *d, int ticks ) {
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
    kernel<<<blocks,threads>>>( d->dev_bitmap, ticks );

    HANDLE_ERROR( hipMemcpy( d->bitmap->get_ptr(),
                              d->dev_bitmap,
                              d->bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );
}

// clean up memory allocated on the GPU
void cleanup( DataBlock *d ) {
    HANDLE_ERROR( hipFree( d->dev_bitmap ) ); 
}

int main( void ) {
    DataBlock   data;
    CPUAnimBitmap  bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&data.dev_bitmap,
                              bitmap.image_size() ) );

    bitmap.anim_and_exit( (void (*)(void*,int))generate_frame,
                            (void (*)(void*))cleanup );
}
