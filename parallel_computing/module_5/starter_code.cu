#include "hip/hip_runtime.h"
//transpose.cu
#include <stdio.h>
#include "gputimer.h"
// #include "utils.h"
#define NumSMPs 30 //Machine dependent see gpu_enum
#define M 8192 //problem size
#define ThreadsperBlock 1024
#define BlocksperGrid M/ThreadsperBlock

const int N= 1024;	// matrix size will be NxN

int compare_matrices(float *gpu, float *ref, int N)
{
        int result = 0;
        for(int j=0; j < N; j++)
        for(int i=0; i < N; i++)
                if (ref[i + j*N] != gpu[i + j*N])
                   {result = 1;}
        return result;
}

// fill a matrix with sequential numbers in the range 0..N-1
void fill_matrix(float *mat, int N)
{
        for(int j=0; j < N * N; j++)
                mat[j] = (float) j;
}


void 
transpose_CPU(float in[], float out[])
{
	for(int j=0; j < N; j++)
    	    for(int i=0; i < N; i++)
      		out[j + i*N] = in[i + j*N]; // implements flip out(j,i) = in(i,j)
}

// to be launched on a single thread
__global__ void 
transpose_serial(float in[], float out[])
{
	for(int j=0; j < N; j++)
		for(int i=0; i < N; i++)
			out[j + i*N] = in[i + j*N]; 
}

// to be launched with one thread per row of output matrix
__global__ void 
transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x + blockDim.x * blockIdx.y;

	for(int j=0; j < N; j++)
		out[j + i*N] = in[i + j*N]; 
}

__global__ void 
transpose_parallel_per_element_tiled(float in[], float out[])
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Column

    if (i < N && j < N) {
        out[i + j*N] = in[j + i*N]; // Transpose
    }
}

__global__ void 
transpose_parallel_per_element_tiled_shared(float in[], float out[])
{
    // Global tiles
    __shared__ float tile[16][16]; // KxK shared memory tile

    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row 
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Column
    
// Load from global memory to shared memory
    if (i < N && j < N) {
        tile[threadIdx.y][threadIdx.x] = in[i* N + j];

    }

    // Write from shared memory to global memory
    if (i < N && j < N){
        out[j * N + i] = tile[threadIdx.y][threadIdx.x];
    }
}

int main(int argc, char **argv)
{
	int numbytes = N * N * sizeof(float);
	float *in = (float *) malloc(numbytes);
	float *out = (float *) malloc(numbytes);
	float *gold = (float *) malloc(numbytes);
	fill_matrix(in, N);
	transpose_CPU(in, gold);

	float *d_in, *d_out;

	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

	GpuTimer timer;
        timer.Start();
	transpose_serial<<<1,1>>>(d_in, d_out);
	timer.Stop();
        for (int i=0; i < N*N; ++i){out[i] = 0.0;}
        hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_serial: %g ms.\nVerifying ...%s\n", 
		   timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

   
        hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
        timer.Start();
	transpose_parallel_per_row<<<1,N>>>(d_in, d_out);
	timer.Stop();
        for (int i=0; i < N*N; ++i){out[i] = 0.0;}  //clean out
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_row: %g ms.\nVerifying ...%s\n", 
		    timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

        hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
        // Tiled versions
        const int K= 16;
        dim3 blocks_tiled(N/K,N/K);
	dim3 threads_tiled(K,K);
	timer.Start();
	transpose_parallel_per_element_tiled<<<blocks_tiled,threads_tiled>>>(d_in, d_out);
	timer.Stop();
        for (int i=0; i < N*N; ++i){out[i] = 0.0;}
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled %dx%d: %g ms.\nVerifying ...%s\n", 
		   K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");
        hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
        dim3 blocks_tiled_sh(N/K,N/K);
	dim3 threads_tiled_sh(K,K);
        timer.Start();
	transpose_parallel_per_element_tiled_shared<<<blocks_tiled_sh,threads_tiled_sh>>>(d_in, d_out);
	timer.Stop();
        for (int i=0; i < N*N; ++i){out[i] = 0.0;}
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled_shared %dx%d: %g ms.\nVerifying ...%s\n", 
		   K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

	hipFree(d_in);
	hipFree(d_out);
}

